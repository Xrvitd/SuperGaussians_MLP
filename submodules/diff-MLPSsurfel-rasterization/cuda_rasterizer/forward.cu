#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
// #include "mlp.cu"

namespace cg = cooperative_groups;





__device__ glm::vec4 MySigmoidf(glm::vec4 x)
{
	return glm::vec4(1.0f / (1.0f + exp(-x.x)), 1.0f / (1.0f + exp(-x.y)), 1.0f / (1.0f + exp(-x.z)), 1.0f / (1.0f + exp(-x.w)));
}
__device__ glm::vec4 MyReluf(glm::vec4 x)
{
	return glm::vec4(x.x > 0.0f? x.x : 0.0f,  x.y > 0.0f? x.y : 0.0f,  x.z > 0.0f? x.z : 0.0f,  x.w > 0.0f? x.w : 0.0f);
}
// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * (34);
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Compute a 2D-to-2D mapping matrix from a tangent plane into a image plane
// given a 2D gaussian parameters.
__device__ void compute_transmat(
	const float3& p_orig,
	const glm::vec2 scale,
	float mod,
	const glm::vec4 rot,
	const float* projmatrix,
	const float* viewmatrix,
	const int W,
	const int H, 
	glm::mat3 &T,
	float3 &normal
) {

	glm::mat3 R = quat_to_rotmat(rot);
	glm::mat3 S = scale_to_mat(scale, mod);
	glm::mat3 L = R * S;

	// center of Gaussians in the camera coordinate
	glm::mat3x4 splat2world = glm::mat3x4(
		glm::vec4(L[0], 0.0),
		glm::vec4(L[1], 0.0),
		glm::vec4(p_orig.x, p_orig.y, p_orig.z, 1)
	);

	glm::mat4 world2ndc = glm::mat4(
		projmatrix[0], projmatrix[4], projmatrix[8], projmatrix[12],
		projmatrix[1], projmatrix[5], projmatrix[9], projmatrix[13],
		projmatrix[2], projmatrix[6], projmatrix[10], projmatrix[14],
		projmatrix[3], projmatrix[7], projmatrix[11], projmatrix[15]
	);

	glm::mat3x4 ndc2pix = glm::mat3x4(
		glm::vec4(float(W) / 2.0, 0.0, 0.0, float(W-1) / 2.0),
		glm::vec4(0.0, float(H) / 2.0, 0.0, float(H-1) / 2.0),
		glm::vec4(0.0, 0.0, 0.0, 1.0)
	);

	T = glm::transpose(splat2world) * world2ndc * ndc2pix;
	normal = transformVec4x3({L[2].x, L[2].y, L[2].z}, viewmatrix);

}

// Computing the bounding box of the 2D Gaussian and its center
// The center of the bounding box is used to create a low pass filter
__device__ bool compute_aabb(
	glm::mat3 T, 
	float cutoff,
	float2& point_image,
	float2& extent
) {
	glm::vec3 t = glm::vec3(cutoff * cutoff, cutoff * cutoff, -1.0f);
	float d = glm::dot(t, T[2] * T[2]);
	if (d == 0.0) return false;
	glm::vec3 f = (1 / d) * t;

	glm::vec2 p = glm::vec2(
		glm::dot(f, T[0] * T[2]),
		glm::dot(f, T[1] * T[2])
	);

	glm::vec2 h0 = p * p - 
		glm::vec2(
			glm::dot(f, T[0] * T[0]),
			glm::dot(f, T[1] * T[1])
		);

	glm::vec2 h = sqrt(max(glm::vec2(1e-4, 1e-4), h0));
	point_image = {p.x, p.y};
	extent = {h.x, h.y};
	return true;
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec2* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* transMat_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, const float tan_fovy,
	const float focal_x, const float focal_y,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float* transMats,
	float* rgb,
	float4* normal_opacity,
	float4* my_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;
	
	// Compute transformation matrix
	glm::mat3 T;
	float3 normal;
	if (transMat_precomp == nullptr)
	{
		compute_transmat(((float3*)orig_points)[idx], scales[idx], scale_modifier, rotations[idx], projmatrix, viewmatrix, W, H, T, normal);
		float3 *T_ptr = (float3*)transMats;
		T_ptr[idx * 3 + 0] = {T[0][0], T[0][1], T[0][2]};
		T_ptr[idx * 3 + 1] = {T[1][0], T[1][1], T[1][2]};
		T_ptr[idx * 3 + 2] = {T[2][0], T[2][1], T[2][2]};
	} else {
		glm::vec3 *T_ptr = (glm::vec3*)transMat_precomp;
		T = glm::mat3(
			T_ptr[idx * 3 + 0], 
			T_ptr[idx * 3 + 1],
			T_ptr[idx * 3 + 2]
		);
		normal = make_float3(0.0, 0.0, 1.0);
	}

#if DUAL_VISIABLE
	float cos = -sumf3(p_view * normal);
	if (cos == 0) return;
	float multiplier = cos > 0 ? 1: -1;
	normal = multiplier * normal;
#endif

#if TIGHTBBOX // no use in the paper, but it indeed help speeds.
	// the effective extent is now depended on the opacity of gaussian.
	float cutoff = sqrtf(max(9.f + 2.f * logf(opacities[idx]), 0.000001));
#else
	float cutoff = 3.0f;
#endif

	// Compute center and radius
	float2 point_image;
	float radius;
	{
		float2 extent;
		bool ok = compute_aabb(T, cutoff, point_image, extent);
		if (!ok) return;
		radius = ceil(max(max(extent.x, extent.y), cutoff * FilterSize));
	}

	uint2 rect_min, rect_max;
	getRect(point_image, radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// Compute colors 
	if (colors_precomp == nullptr) {
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	depths[idx] = p_view.z;
	radii[idx] = (int)radius;
	points_xy_image[idx] = point_image;
	normal_opacity[idx] = {normal.x, normal.y, normal.z, opacities[idx*4]};
	my_opacity[idx] = {opacities[idx*4 + 0], opacities[idx*4 + 1], opacities[idx*4 + 2], opacities[idx*4 + 3]};
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float* __restrict__ transMats,
	const float* __restrict__ depths,
	const float4* __restrict__ normal_opacity,
	const float4* __restrict__ my_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	float* __restrict__ out_others,
	const float* __restrict__ shs,
	float2* __restrict__ wichzone)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y};

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_normal_opacity[BLOCK_SIZE];
	__shared__ float4 collected_my_opacity[BLOCK_SIZE];

	__shared__ float3 collected_Tu[BLOCK_SIZE];
	__shared__ float3 collected_Tv[BLOCK_SIZE];
	__shared__ float3 collected_Tw[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };


#if RENDER_AXUTILITY
	// render axutility ouput
	float N[3] = {0};
	float D = { 0 };
	float M1 = {0};
	float M2 = {0};
	float distortion = {0};
	float median_depth = {0};
	// float median_weight = {0};
	float median_contributor = {-1};

#endif

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_normal_opacity[block.thread_rank()] = normal_opacity[coll_id];
			collected_my_opacity[block.thread_rank()] = my_opacity[coll_id];
			collected_Tu[block.thread_rank()] = {transMats[9 * coll_id+0], transMats[9 * coll_id+1], transMats[9 * coll_id+2]};
			collected_Tv[block.thread_rank()] = {transMats[9 * coll_id+3], transMats[9 * coll_id+4], transMats[9 * coll_id+5]};
			collected_Tw[block.thread_rank()] = {transMats[9 * coll_id+6], transMats[9 * coll_id+7], transMats[9 * coll_id+8]};
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Fisrt compute two homogeneous planes, See Eq. (8)
			const float2 xy = collected_xy[j];
			const float3 Tu = collected_Tu[j];
			const float3 Tv = collected_Tv[j];
			const float3 Tw = collected_Tw[j];

			float3 k = pix.x * Tw - Tu;
			float3 l = pix.y * Tw - Tv;
			float3 p = cross(k, l);
			if (p.z == 0.0) continue;
			float2 s = {p.x / p.z, p.y / p.z};
			float rho3d = (s.x * s.x + s.y * s.y); 
			float2 d = {xy.x - pixf.x, xy.y - pixf.y};
			float rho2d = FilterInvSquare * (d.x * d.x + d.y * d.y); 

			// compute intersection and depth
			float rho = min(rho3d, rho2d);
			float depth = (rho3d <= rho2d) ? (s.x * Tw.x + s.y * Tw.y) + Tw.z : Tw.z; 
			if (depth < near_n) continue;
			float4 nor_o = collected_normal_opacity[j];
			float4 my_o = collected_my_opacity[j];
			float normal[3] = {nor_o.x, nor_o.y, nor_o.z};
			// float opa = nor_o.w; // old 2dgs


			float proj_u = s.x;
			float proj_v = s.y;
			if(rho3d>rho2d)
			{
				proj_u = d.x;
				proj_v = d.y;
			}

			glm::vec3* sh = ((glm::vec3*)shs) + collected_id[j] * (16+18);

			glm::mat2x4 input = glm::mat2x4(
				glm::vec4(sh[16+0].x, sh[16+0].y, sh[16+0].z, sh[16+1].x),
				glm::vec4(sh[16+1].y, sh[16+1].z, sh[16+2].x, sh[16+2].y)
			);
			glm::vec4 input_biases = glm::vec4(sh[16+2].z, sh[16+3].x, sh[16+3].y, sh[16+3].z);
			glm::mat4x4 middle = glm::mat4x4(
				glm::vec4(sh[16+4].x, sh[16+4].y, sh[16+4].z, sh[16+5].x),
				glm::vec4(sh[16+5].y, sh[16+5].z, sh[16+6].x, sh[16+6].y),
				glm::vec4(sh[16+6].z, sh[16+7].x, sh[16+7].y, sh[16+7].z),
				glm::vec4(sh[16+8].x, sh[16+8].y, sh[16+8].z, sh[16+9].x)
			);
			glm::vec4 middle_biases = glm::vec4(sh[16+9].y, sh[16+9].z, sh[16+10].x, sh[16+10].y);
			glm::mat4x4 output = glm::mat4x4(
				glm::vec4(sh[16+10].z, sh[16+11].x, sh[16+11].y, sh[16+11].z),
				glm::vec4(sh[16+12].x, sh[16+12].y, sh[16+12].z, sh[16+13].x),
				glm::vec4(sh[16+13].y, sh[16+13].z, sh[16+14].x, sh[16+14].y),
				glm::vec4(sh[16+14].z, sh[16+15].x, sh[16+15].y, sh[16+15].z)
			);
			glm::vec4 output_biases = glm::vec4(sh[16+16].x, sh[16+16].y, sh[16+16].z, sh[16+17].x);

			// forward
			glm::vec2 input_data2 = glm::vec2(proj_u, proj_v);
			input_data2.x = 1.0f / (1.0f + exp(-proj_u));
			input_data2.y = 1.0f / (1.0f + exp(-proj_v));


			glm::vec4 L1out = input  * input_data2  + input_biases;
			glm::vec4 L1outR = MySigmoidf(L1out);
			glm::vec4 L2out = middle * L1outR + middle_biases;
			glm::vec4 L2outR = MySigmoidf(L2out);
			glm::vec4 L3out = output * L2outR  + output_biases;
			glm::vec4 L3outR = MySigmoidf(L3out);
			// float x[3] = {L3outR.x, L3outR.y, L3outR.z};
			float x[3] = {L3outR.x*2.0f -1.0f, L3outR.y*2.0f -1.0f, L3outR.z*2.0f -1.0f};

			// if(!isnan(MLPopa[collected_id[j]]))
			// {
			// 	MLPopa[collected_id[j]] = max(MLPopa[collected_id[j]], L3outR.w);
			// }else{
			// 	MLPopa[collected_id[j]] = L3outR.w;
			// }
			// MLPopa[collected_id[j]] = L3outR.w;

			// float my_opa = my_o.x;
			float my_opa = my_o.x + 0.5 * (L3outR.w * 2.0 - 1.0);
			// float my_opa = L3outR.w;
			// printf("my_rho: %f, %f, %f, %f | my_o: %f, %f, %f, %f | my_opa: %f\n", my_rho[0], my_rho[1], my_rho[2], my_rho[3], my_o.x, my_o.y, my_o.z, my_o.w, my_opa);

			float power = -0.5f * rho;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, my_opa * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			} 
			// if(rho3d < rho2d && (s.x>3.0 || s.y>3.0)) 
			// 	printf("s: %f, %f\n", s.x, s.y);

			float w = alpha * T;
#if RENDER_AXUTILITY
			// Render depth distortion map
			// Efficient implementation of distortion loss, see 2DGS' paper appendix.
			float A = 1-T;
			float m = far_n / (far_n - near_n) * (1 - near_n / depth);
			distortion += (m * m * A + M2 - 2 * m * M1) * w;
			D  += depth * w;
			M1 += m * w;
			M2 += m * m * w;

			if (T > 0.5) {
				median_depth = depth;
				// median_weight = w;
				median_contributor = contributor;
			}
			// Render normal map
			for (int ch=0; ch<3; ch++) N[ch] += normal[ch] * w;
#endif




			// mlp
			// double lr= 0.004, beta = 0.35;
			

			// printf("x: %f, %f, %f\n", x[0], x[1], x[2]);

			
			





			// end mlp


			float Myalpha = 1.0;
			float nncolor[3];
			nncolor[0] = x[0];
			nncolor[1] = x[1];
			nncolor[2] = x[2];
			// printf("nncolor: %f, %f, %f\n", nncolor[0], nncolor[1], nncolor[2]);

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
			{
				// C[ch] += features[collected_id[j] * CHANNELS + ch] * w;
				C[ch] += (features[collected_id[j] * CHANNELS + ch] + Myalpha * nncolor[ch]) * w;
				// C[ch] += (Myalpha * nncolor[ch]) * w;
			}
				
			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];

#if RENDER_AXUTILITY
		n_contrib[pix_id + H * W] = median_contributor;
		final_T[pix_id + H * W] = M1;
		final_T[pix_id + 2 * H * W] = M2;
		out_others[pix_id + DEPTH_OFFSET * H * W] = D;
		out_others[pix_id + ALPHA_OFFSET * H * W] = 1 - T;
		for (int ch=0; ch<3; ch++) out_others[pix_id + (NORMAL_OFFSET+ch) * H * W] = N[ch];
		out_others[pix_id + MIDDEPTH_OFFSET * H * W] = median_depth;
		out_others[pix_id + DISTORTION_OFFSET * H * W] = distortion;
		// out_others[pix_id + MEDIAN_WEIGHT_OFFSET * H * W] = median_weight;
#endif
	}
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float2* means2D,
	const float* colors,
	const float* transMats,
	const float* depths,
	const float4* normal_opacity,
	const float4* my_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	float* out_others,
	const float* shs,
	float2* wichzone)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		focal_x, focal_y,
		means2D,
		colors,
		transMats,
		depths,
		normal_opacity,
		my_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color,
		out_others,
		shs,
		wichzone);
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec2* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* transMat_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, const int H,
	const float focal_x, const float focal_y,
	const float tan_fovx, const float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* transMats,
	float* rgb,
	float4* normal_opacity,
	float4* my_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		transMat_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		transMats,
		rgb,
		normal_opacity,
		my_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}
