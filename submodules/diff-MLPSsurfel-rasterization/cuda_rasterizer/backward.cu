#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "backward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
// #include "mlp.h"
// #include "common.h"
namespace cg = cooperative_groups;


__device__ glm::vec4 MySigmoid(glm::vec4 x)
{
	return glm::vec4(1.0f / (1.0f + exp(-x.x)), 1.0f / (1.0f + exp(-x.y)), 1.0f / (1.0f + exp(-x.z)), 1.0f / (1.0f + exp(-x.w)));
}

__device__ glm::vec4 dL_dSigmoid(glm::vec4 x)
{
	return glm::vec4(1.0f / (1.0f + exp(-x.x)) * (1.0f - 1.0f / (1.0f + exp(-x.x))), 
					 1.0f / (1.0f + exp(-x.y)) * (1.0f - 1.0f / (1.0f + exp(-x.y))), 
					 1.0f / (1.0f + exp(-x.z)) * (1.0f - 1.0f / (1.0f + exp(-x.z))), 
					 1.0f / (1.0f + exp(-x.w)) * (1.0f - 1.0f / (1.0f + exp(-x.w))));
}
__device__ glm::vec4 MyRelu(glm::vec4 x)
{
	return glm::vec4(x.x > 0.0f? x.x : 0.0f,  x.y > 0.0f? x.y : 0.0f,  x.z > 0.0f? x.z : 0.0f,  x.w > 0.0f? x.w : 0.0f);
}
__device__ glm::vec4 dL_dRelu(glm::vec4 x)
{
	return glm::vec4(x.x > 0.0f? 1.0f : 0.0f,  x.y > 0.0f? 1.0f : 0.0f,  x.z > 0.0f? 1.0f : 0.0f,  x.w > 0.0f? 1.0f : 0.0f);
}

// Backward pass for conversion of spherical harmonics to RGB for
// each Gaussian.
__device__ void computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, const bool* clamped, const glm::vec3* dL_dcolor, glm::vec3* dL_dmeans, glm::vec3* dL_dshs, float2* wichzone)
{
	// Compute intermediate values, as it is done during forward
	glm::vec3 pos = means[idx];
	glm::vec3 dir_orig = pos - campos;
	glm::vec3 dir = dir_orig / glm::length(dir_orig);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * (34);

	// Use PyTorch rule for clamping: if clamping was applied,
	// gradient becomes 0.
	glm::vec3 dL_dRGB = dL_dcolor[idx];
	dL_dRGB.x *= clamped[3 * idx + 0] ? 0 : 1;
	dL_dRGB.y *= clamped[3 * idx + 1] ? 0 : 1;
	dL_dRGB.z *= clamped[3 * idx + 2] ? 0 : 1;

	glm::vec3 dRGBdx(0, 0, 0);
	glm::vec3 dRGBdy(0, 0, 0);
	glm::vec3 dRGBdz(0, 0, 0);
	float x = dir.x;
	float y = dir.y;
	float z = dir.z;

	// Target location for this Gaussian to write SH gradients to
	glm::vec3* dL_dsh = dL_dshs + idx * (34);

	// float proj_u = wichzone[idx].x;
	// float proj_v = wichzone[idx].y;
	// // printf("Backward proj_u: %f, proj_v: %f\n", proj_u, proj_v);
	// if(isnan(proj_u) || isnan(proj_v))
	// {
	// 	printf("Backward proj_u: %f, proj_v: %f\n", proj_u, proj_v);
	// 	proj_u = 0.5;
	// 	proj_v = 0.5;
	// }
	// dL_dsh[16] = (1.0f-proj_u) * (1.0f - proj_v) * dL_dRGB;
	// dL_dsh[17] = (1.0f-proj_u) * proj_v * dL_dRGB;
	// dL_dsh[18] = proj_u * (1.0f - proj_v) * dL_dRGB;
	// dL_dsh[19] = proj_u * proj_v * dL_dRGB;
	// dL_dsh[16] =  dL_dRGB;
	// dL_dsh[17] = dL_dRGB;
	// dL_dsh[18] =  dL_dRGB;
	// dL_dsh[19] =dL_dRGB;
	// printf("Backward proj_u: %f, proj_v: %f | dL_dRGB: %f, %f, %f\n", proj_u, proj_v, dL_dRGB.x, dL_dRGB.y, dL_dRGB.z);


	// No tricks here, just high school-level calculus.
	float dRGBdsh0 = SH_C0;
	dL_dsh[0] = dRGBdsh0 * dL_dRGB;
	if (deg > 0)
	{
		float dRGBdsh1 = -SH_C1 * y;
		float dRGBdsh2 = SH_C1 * z;
		float dRGBdsh3 = -SH_C1 * x;
		dL_dsh[1] = dRGBdsh1 * dL_dRGB;
		dL_dsh[2] = dRGBdsh2 * dL_dRGB;
		dL_dsh[3] = dRGBdsh3 * dL_dRGB;

		dRGBdx = -SH_C1 * sh[3];
		dRGBdy = -SH_C1 * sh[1];
		dRGBdz = SH_C1 * sh[2];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;

			float dRGBdsh4 = SH_C2[0] * xy;
			float dRGBdsh5 = SH_C2[1] * yz;
			float dRGBdsh6 = SH_C2[2] * (2.f * zz - xx - yy);
			float dRGBdsh7 = SH_C2[3] * xz;
			float dRGBdsh8 = SH_C2[4] * (xx - yy);
			dL_dsh[4] = dRGBdsh4 * dL_dRGB;
			dL_dsh[5] = dRGBdsh5 * dL_dRGB;
			dL_dsh[6] = dRGBdsh6 * dL_dRGB;
			dL_dsh[7] = dRGBdsh7 * dL_dRGB;
			dL_dsh[8] = dRGBdsh8 * dL_dRGB;

			dRGBdx += SH_C2[0] * y * sh[4] + SH_C2[2] * 2.f * -x * sh[6] + SH_C2[3] * z * sh[7] + SH_C2[4] * 2.f * x * sh[8];
			dRGBdy += SH_C2[0] * x * sh[4] + SH_C2[1] * z * sh[5] + SH_C2[2] * 2.f * -y * sh[6] + SH_C2[4] * 2.f * -y * sh[8];
			dRGBdz += SH_C2[1] * y * sh[5] + SH_C2[2] * 2.f * 2.f * z * sh[6] + SH_C2[3] * x * sh[7];

			if (deg > 2)
			{
				float dRGBdsh9 = SH_C3[0] * y * (3.f * xx - yy);
				float dRGBdsh10 = SH_C3[1] * xy * z;
				float dRGBdsh11 = SH_C3[2] * y * (4.f * zz - xx - yy);
				float dRGBdsh12 = SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy);
				float dRGBdsh13 = SH_C3[4] * x * (4.f * zz - xx - yy);
				float dRGBdsh14 = SH_C3[5] * z * (xx - yy);
				float dRGBdsh15 = SH_C3[6] * x * (xx - 3.f * yy);
				dL_dsh[9] = dRGBdsh9 * dL_dRGB;
				dL_dsh[10] = dRGBdsh10 * dL_dRGB;
				dL_dsh[11] = dRGBdsh11 * dL_dRGB;
				dL_dsh[12] = dRGBdsh12 * dL_dRGB;
				dL_dsh[13] = dRGBdsh13 * dL_dRGB;
				dL_dsh[14] = dRGBdsh14 * dL_dRGB;
				dL_dsh[15] = dRGBdsh15 * dL_dRGB;

				dRGBdx += (
					SH_C3[0] * sh[9] * 3.f * 2.f * xy +
					SH_C3[1] * sh[10] * yz +
					SH_C3[2] * sh[11] * -2.f * xy +
					SH_C3[3] * sh[12] * -3.f * 2.f * xz +
					SH_C3[4] * sh[13] * (-3.f * xx + 4.f * zz - yy) +
					SH_C3[5] * sh[14] * 2.f * xz +
					SH_C3[6] * sh[15] * 3.f * (xx - yy));

				dRGBdy += (
					SH_C3[0] * sh[9] * 3.f * (xx - yy) +
					SH_C3[1] * sh[10] * xz +
					SH_C3[2] * sh[11] * (-3.f * yy + 4.f * zz - xx) +
					SH_C3[3] * sh[12] * -3.f * 2.f * yz +
					SH_C3[4] * sh[13] * -2.f * xy +
					SH_C3[5] * sh[14] * -2.f * yz +
					SH_C3[6] * sh[15] * -3.f * 2.f * xy);

				dRGBdz += (
					SH_C3[1] * sh[10] * xy +
					SH_C3[2] * sh[11] * 4.f * 2.f * yz +
					SH_C3[3] * sh[12] * 3.f * (2.f * zz - xx - yy) +
					SH_C3[4] * sh[13] * 4.f * 2.f * xz +
					SH_C3[5] * sh[14] * (xx - yy));
			}
		}
	}

	// The view direction is an input to the computation. View direction
	// is influenced by the Gaussian's mean, so SHs gradients
	// must propagate back into 3D position.
	glm::vec3 dL_ddir(glm::dot(dRGBdx, dL_dRGB), glm::dot(dRGBdy, dL_dRGB), glm::dot(dRGBdz, dL_dRGB));

	// Account for normalization of direction
	float3 dL_dmean = dnormvdv(float3{ dir_orig.x, dir_orig.y, dir_orig.z }, float3{ dL_ddir.x, dL_ddir.y, dL_ddir.z });

	// Gradients of loss w.r.t. Gaussian means, but only the portion 
	// that is caused because the mean affects the view-dependent color.
	// Additional mean gradient is accumulated in below methods.
	dL_dmeans[idx] += glm::vec3(dL_dmean.x, dL_dmean.y, dL_dmean.z);
}


// Backward version of the rendering procedure.
template <uint32_t C>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float* __restrict__ bg_color,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ normal_opacity,
	const float4* __restrict__ my_opacity,
	const float* __restrict__ transMats,
	const float* __restrict__ colors,
	const float* __restrict__ depths,
	const float* __restrict__ final_Ts,
	const uint32_t* __restrict__ n_contrib,
	const float* __restrict__ dL_dpixels,
	const float* __restrict__ dL_depths,
	float * __restrict__ dL_dtransMat,
	float3* __restrict__ dL_dmean2D,
	float* __restrict__ dL_dnormal3D,
	float* __restrict__ dL_dopacity,
	float* __restrict__ dL_dcolors,
	const float* __restrict__ shs,
	float2* __restrict__ wichzone,
	float* __restrict__ dL_dshs)
{
	// We rasterize again. Compute necessary block info.
	auto block = cg::this_thread_block();
	const uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	const uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	const uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	const uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	const uint32_t pix_id = W * pix.y + pix.x;
	const float2 pixf = {(float)pix.x, (float)pix.y};

	const bool inside = pix.x < W&& pix.y < H;
	const uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];

	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);

	bool done = !inside;
	int toDo = range.y - range.x;

	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_normal_opacity[BLOCK_SIZE];
	__shared__ float4 collected_my_opacity[BLOCK_SIZE];
	__shared__ float collected_colors[C * BLOCK_SIZE];
	__shared__ float3 collected_Tu[BLOCK_SIZE];
	__shared__ float3 collected_Tv[BLOCK_SIZE];
	__shared__ float3 collected_Tw[BLOCK_SIZE];
	// __shared__ float collected_depths[BLOCK_SIZE];

	// In the forward, we stored the final value for T, the
	// product of all (1 - alpha) factors. 
	const float T_final = inside ? final_Ts[pix_id] : 0;
	float T = T_final;

	// We start from the back. The ID of the last contributing
	// Gaussian is known from each pixel from the forward.
	uint32_t contributor = toDo;
	const int last_contributor = inside ? n_contrib[pix_id] : 0;

	float accum_rec[C] = { 0 };
	float dL_dpixel[C];

#if RENDER_AXUTILITY
	float dL_dreg;
	float dL_ddepth;
	float dL_daccum;
	float dL_dnormal2D[3];
	const int median_contributor = inside ? n_contrib[pix_id + H * W] : 0;
	float dL_dmedian_depth;
	float dL_dmax_dweight;

	if (inside) {
		dL_ddepth = dL_depths[DEPTH_OFFSET * H * W + pix_id];
		dL_daccum = dL_depths[ALPHA_OFFSET * H * W + pix_id];
		dL_dreg = dL_depths[DISTORTION_OFFSET * H * W + pix_id];
		for (int i = 0; i < 3; i++) 
			dL_dnormal2D[i] = dL_depths[(NORMAL_OFFSET + i) * H * W + pix_id];

		dL_dmedian_depth = dL_depths[MIDDEPTH_OFFSET * H * W + pix_id];
		// dL_dmax_dweight = dL_depths[MEDIAN_WEIGHT_OFFSET * H * W + pix_id];
	}

	// for compute gradient with respect to depth and normal
	float last_depth = 0;
	float last_normal[3] = { 0 };
	float accum_depth_rec = 0;
	float accum_alpha_rec = 0;
	float accum_normal_rec[3] = {0};
	// for compute gradient with respect to the distortion map
	const float final_D = inside ? final_Ts[pix_id + H * W] : 0;
	const float final_D2 = inside ? final_Ts[pix_id + 2 * H * W] : 0;
	const float final_A = 1 - T_final;
	float last_dL_dT = 0;
#endif

	if (inside){
		for (int i = 0; i < C; i++)
			dL_dpixel[i] = dL_dpixels[i * H * W + pix_id];
	}

	float last_alpha = 0;
	float last_color[C] = { 0 };

	// Gradient of pixel coordinate w.r.t. normalized 
	// screen-space viewport corrdinates (-1 to 1)
	const float ddelx_dx = 0.5 * W;
	const float ddely_dy = 0.5 * H;

	// Traverse all Gaussians
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// Load auxiliary data into shared memory, start in the BACK
		// and load them in revers order.
		block.sync();
		const int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			const int coll_id = point_list[range.y - progress - 1];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_normal_opacity[block.thread_rank()] = normal_opacity[coll_id];
			collected_my_opacity[block.thread_rank()] = my_opacity[coll_id];
			collected_Tu[block.thread_rank()] = {transMats[9 * coll_id+0], transMats[9 * coll_id+1], transMats[9 * coll_id+2]};
			collected_Tv[block.thread_rank()] = {transMats[9 * coll_id+3], transMats[9 * coll_id+4], transMats[9 * coll_id+5]};
			collected_Tw[block.thread_rank()] = {transMats[9 * coll_id+6], transMats[9 * coll_id+7], transMats[9 * coll_id+8]};
			for (int i = 0; i < C; i++)
				collected_colors[i * BLOCK_SIZE + block.thread_rank()] = colors[coll_id * C + i];
				// collected_depths[block.thread_rank()] = depths[coll_id];
		}
		block.sync();

		// Iterate over Gaussians
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current Gaussian ID. Skip, if this one
			// is behind the last contributor for this pixel.
			contributor--;
			if (contributor >= last_contributor)
				continue;

			// compute ray-splat intersection as before
			// Fisrt compute two homogeneous planes, See Eq. (8)
			const float2 xy = collected_xy[j];
			const float3 Tu = collected_Tu[j];
			const float3 Tv = collected_Tv[j];
			const float3 Tw = collected_Tw[j];
			float3 k = pix.x * Tw - Tu;
			float3 l = pix.y * Tw - Tv;
			float3 p = cross(k, l);
			if (p.z == 0.0) continue;
			float2 s = {p.x / p.z, p.y / p.z};
			float rho3d = (s.x * s.x + s.y * s.y); 
			float2 d = {xy.x - pixf.x, xy.y - pixf.y};
			float rho2d = FilterInvSquare * (d.x * d.x + d.y * d.y); 

			// compute intersection and depth
			float rho = min(rho3d, rho2d);
			float c_d = (rho3d <= rho2d) ? (s.x * Tw.x + s.y * Tw.y) + Tw.z : Tw.z; 
			if (c_d < near_n) continue;
			float4 nor_o = collected_normal_opacity[j];
			float4 my_o = collected_my_opacity[j];
			float normal[3] = {nor_o.x, nor_o.y, nor_o.z};
			// float opa = nor_o.w; // old 2dgs
			float proj_u = s.x;
			float proj_v = s.y;
			if(rho3d>rho2d)
			{
				proj_u = d.x;
				proj_v = d.y;
			}

			glm::vec3* sh = ((glm::vec3*)shs) + collected_id[j] * (16+18);

			glm::mat2x4 input = glm::mat2x4(
				glm::vec4(sh[16+0].x, sh[16+0].y, sh[16+0].z, sh[16+1].x),
				glm::vec4(sh[16+1].y, sh[16+1].z, sh[16+2].x, sh[16+2].y)
			);
			glm::vec4 input_biases = glm::vec4(sh[16+2].z, sh[16+3].x, sh[16+3].y, sh[16+3].z);
			glm::mat4x4 middle = glm::mat4x4(
				glm::vec4(sh[16+4].x, sh[16+4].y, sh[16+4].z, sh[16+5].x),
				glm::vec4(sh[16+5].y, sh[16+5].z, sh[16+6].x, sh[16+6].y),
				glm::vec4(sh[16+6].z, sh[16+7].x, sh[16+7].y, sh[16+7].z),
				glm::vec4(sh[16+8].x, sh[16+8].y, sh[16+8].z, sh[16+9].x)
			);
			glm::vec4 middle_biases = glm::vec4(sh[16+9].y, sh[16+9].z, sh[16+10].x, sh[16+10].y);
			glm::mat4x4 output = glm::mat4x4(
				glm::vec4(sh[16+10].z, sh[16+11].x, sh[16+11].y, sh[16+11].z),
				glm::vec4(sh[16+12].x, sh[16+12].y, sh[16+12].z, sh[16+13].x),
				glm::vec4(sh[16+13].y, sh[16+13].z, sh[16+14].x, sh[16+14].y),
				glm::vec4(sh[16+14].z, sh[16+15].x, sh[16+15].y, sh[16+15].z)
			);
			glm::vec4 output_biases = glm::vec4(sh[16+16].x, sh[16+16].y, sh[16+16].z, sh[16+17].x);

			// forward
			glm::vec2 input_data2 = glm::vec2(proj_u, proj_v);
			input_data2.x = 1.0f / (1.0f + exp(-proj_u));
			input_data2.y = 1.0f / (1.0f + exp(-proj_v));

			glm::vec4 L1out = input  * input_data2  + input_biases;
			glm::vec4 L1outR = MySigmoid(L1out);
			glm::vec4 L2out = middle * L1outR + middle_biases;
			glm::vec4 L2outR = MySigmoid(L2out);
			glm::vec4 L3out = output * L2outR  + output_biases;
			glm::vec4 L3outR = MySigmoid(L3out);
			// float x[3] = {L3outR.x, L3outR.y, L3outR.z};
			float x[3] = {L3outR.x*2.0f -1.0f, L3outR.y*2.0f -1.0f, L3outR.z*2.0f -1.0f};


		
			// float my_opa = my_o.x;
			float my_opa = my_o.x + 0.5 * (L3outR.w * 2.0 - 1.0);
			// float my_opa = L3outR.w;
			
			// accumulations

			float power = -0.5f * rho;
			if (power > 0.0f)
				continue;

			const float G = exp(power);
			float dalpha_dgsscale = 0.0f;
			dalpha_dgsscale = my_opa * power * exp(power);
			const float alpha = min(0.99f, my_opa * G);
			if (alpha < 1.0f / 255.0f)
				continue;

			T = T / (1.f - alpha);
			const float dchannel_dcolor = alpha * T;
			const float w = alpha * T;
			// Propagate gradients to per-Gaussian colors and keep
			// gradients w.r.t. alpha (blending factor for a Gaussian/pixel
			// pair).
			float dL_dalpha = 0.0f;
			const int global_id = collected_id[j];
			
			// float proj_u = s.x;
			// float proj_v = s.y;
			
			

			float Myalpha = 1.0;
			float nncolor[3];
			nncolor[0] = x[0];
			nncolor[1] = x[1];
			nncolor[2] = x[2];

			




			float dl_dc[3],dc_dsx[3]={0},dc_dsy[3],dop_dsx=0.0,dop_dsy=0.0;
			float dsx_dtu[3],dsx_dtv[3],dsy_dtu[3],dsy_dtv[3],dsx_dtw[3],dsy_dtw[3];


			dsx_dtu[0] = p.x * (-1.0) / (p.z * p.z) * l.y * (-1.0);
			dsx_dtu[1] = (-1.0) / (p.z * p.z) * l.x * p.x + l.z*(-1.0) / p.z;
			dsx_dtu[2] = l.y / p.z;
			dsx_dtv[0] = p.x * (-1.0) / (p.z * p.z) * k.y;
			dsx_dtv[1] = (-1.0) / (p.z * p.z) * k.x *(-1.0) * p.x + k.z / p.z;
			dsx_dtv[2] = k.y*(-1.0) / p.z;
			dsy_dtu[0] = (-1.0) / (p.z * p.z) * l.y *(-1.0) * p.y + l.z / p.z;
			dsy_dtu[1] = p.y * (-1.0) / (p.z * p.z) * l.x;
			dsy_dtu[2] = l.y*(-1.0) / p.z;
			dsy_dtv[0] = (-1.0) / (p.z * p.z) * k.y * p.y + k.z * (-1.0) / p.z;
			dsy_dtv[1] = p.y * (-1.0) / (p.z * p.z) * k.x * (-1.0);
			dsy_dtv[2] = k.x / p.z;
			dsx_dtw[0] = p.x * (-1.0) / (p.z * p.z) * (pix.x*pix.y*Tw.y - pix.x*Tv.y - pix.x*pix.y*Tw.y + pix.y*Tu.y);
			dsx_dtw[1] = (pix.x*pix.y*Tw.z - pix.x*Tv.z - pix.x*pix.y*Tw.z + pix.y*Tu.z) / p.z + (p.x*(-1.0)/(p.z*p.z) * (pix.x*pix.y*Tw.x - pix.y*Tu.x - pix.x*pix.y*Tw.x + pix.x*Tv.x));
			dsx_dtw[2] = (pix.x*pix.y*Tw.y - pix.y*Tu.y - pix.x*pix.y*Tw.y + pix.x*Tv.y) / p.z;

			dsy_dtw[0] = (pix.x*pix.y*Tw.z - pix.y*Tu.z - pix.x*pix.y*Tw.z + pix.x*Tv.z) / p.z + (p.y*(-1.0)/(p.z*p.z) * (pix.x*pix.y*Tw.y - pix.x*Tv.y - pix.x*pix.y*Tw.y + pix.y*Tu.y));
			dsy_dtw[1] = (p.y*(-1.0)/(p.z*p.z) * (pix.x*pix.y*Tw.x - pix.y*Tu.x - pix.x*pix.y*Tw.x + pix.x*Tv.x));
			dsy_dtw[2] = (pix.x*pix.y*Tw.x - pix.x*Tv.x - pix.x*pix.y*Tw.x + pix.y*Tu.x) / p.z;
			float dl_dtu[3],dl_dtv[3],dl_dtw[3];
			for (int ch = 0; ch < C; ch++)
			{
				dl_dtu[ch] = 0.0;
				dl_dtv[ch] = 0.0;
				dl_dtw[ch] = 0.0;
			}

			for (int ch = 0; ch < C; ch++)
			{
				// const float c = collected_colors[ch * BLOCK_SIZE + j];
				const float c = collected_colors[ch * BLOCK_SIZE + j] + Myalpha * nncolor[ch];
				// const float c = Myalpha * nncolor[ch];
				// Update last color (to be used in the next iteration)
				accum_rec[ch] = last_alpha * last_color[ch] + (1.f - last_alpha) * accum_rec[ch];
				last_color[ch] = c;

				const float dL_dchannel = dL_dpixel[ch];
				dL_dalpha += (c - accum_rec[ch]) * dL_dchannel;
				// Update the gradients w.r.t. color of the Gaussian. 
				// Atomic, since this pixel is just one of potentially
				// many that were affected by this Gaussian.
				atomicAdd(&(dL_dcolors[global_id * C + ch]), dchannel_dcolor * dL_dchannel);

				dl_dc[ch] = dchannel_dcolor * dL_dchannel;
	
			}
		
			
		
			

			float dL_dz = 0.0f;
			float dL_dweight = 0;
#if RENDER_AXUTILITY
			const float m_d = far_n / (far_n - near_n) * (1 - near_n / c_d);
			const float dmd_dd = (far_n * near_n) / ((far_n - near_n) * c_d * c_d);
			if (contributor == median_contributor-1) {
				dL_dz += dL_dmedian_depth;
				// dL_dweight += dL_dmax_dweight;
			}
#if DETACH_WEIGHT 
			// if not detached weight, sometimes 
			// it will bia toward creating extragated 2D Gaussians near front
			dL_dweight += 0;
#else
			dL_dweight += (final_D2 + m_d * m_d * final_A - 2 * m_d * final_D) * dL_dreg;
#endif
			dL_dalpha += dL_dweight - last_dL_dT;
			// propagate the current weight W_{i} to next weight W_{i-1}
			last_dL_dT = dL_dweight * alpha + (1 - alpha) * last_dL_dT;
			const float dL_dmd = 2.0f * (T * alpha) * (m_d * final_A - final_D) * dL_dreg;
			dL_dz += dL_dmd * dmd_dd;

			// Propagate gradients w.r.t ray-splat depths
			accum_depth_rec = last_alpha * last_depth + (1.f - last_alpha) * accum_depth_rec;
			last_depth = c_d;
			dL_dalpha += (c_d - accum_depth_rec) * dL_ddepth;
			// Propagate gradients w.r.t. color ray-splat alphas
			accum_alpha_rec = last_alpha * 1.0 + (1.f - last_alpha) * accum_alpha_rec;
			dL_dalpha += (1 - accum_alpha_rec) * dL_daccum;

			// Propagate gradients to per-Gaussian normals
			for (int ch = 0; ch < 3; ch++) {
				accum_normal_rec[ch] = last_alpha * last_normal[ch] + (1.f - last_alpha) * accum_normal_rec[ch];
				last_normal[ch] = normal[ch];
				dL_dalpha += (normal[ch] - accum_normal_rec[ch]) * dL_dnormal2D[ch];
				atomicAdd((&dL_dnormal3D[global_id * 3 + ch]), alpha * T * dL_dnormal2D[ch]);
			}
#endif

			dL_dalpha *= T;
			// Update last alpha (to be used in the next iteration)
			last_alpha = alpha;

			// Account for fact that alpha also influences how much of
			// the background color is added if nothing left to blend
			float bg_dot_dpixel = 0;
			for (int i = 0; i < C; i++)
				bg_dot_dpixel += bg_color[i] * dL_dpixel[i];
			dL_dalpha += (-T_final / (1.f - alpha)) * bg_dot_dpixel;


			// Helpful reusable temporary variables
			const float dL_dG = nor_o.w * dL_dalpha;
#if RENDER_AXUTILITY
			dL_dz += alpha * T * dL_ddepth; 
#endif

			if (rho3d <= rho2d) {
				// Update gradients w.r.t. covariance of Gaussian 3x3 (T)
				const float2 dL_ds = {
					dL_dG * -G * s.x + dL_dz * Tw.x,
					dL_dG * -G * s.y + dL_dz * Tw.y
				};
				const float3 dz_dTw = {s.x, s.y, 1.0};
				const float dsx_pz = dL_ds.x / p.z;
				const float dsy_pz = dL_ds.y / p.z;
				const float3 dL_dp = {dsx_pz, dsy_pz, -(dsx_pz * s.x + dsy_pz * s.y)};
				const float3 dL_dk = cross(l, dL_dp);
				const float3 dL_dl = cross(dL_dp, k);

				const float3 dL_dTu = {-dL_dk.x, -dL_dk.y, -dL_dk.z};
				const float3 dL_dTv = {-dL_dl.x, -dL_dl.y, -dL_dl.z};
				const float3 dL_dTw = {
					pixf.x * dL_dk.x + pixf.y * dL_dl.x + dL_dz * dz_dTw.x, 
					pixf.x * dL_dk.y + pixf.y * dL_dl.y + dL_dz * dz_dTw.y, 
					pixf.x * dL_dk.z + pixf.y * dL_dl.z + dL_dz * dz_dTw.z};


				// Update gradients w.r.t. 3D covariance (3x3 matrix)
				atomicAdd(&dL_dtransMat[global_id * 9 + 0],  dL_dTu.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 1],  dL_dTu.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 2],  dL_dTu.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 3],  dL_dTv.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 4],  dL_dTv.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 5],  dL_dTv.z);
				atomicAdd(&dL_dtransMat[global_id * 9 + 6],  dL_dTw.x);
				atomicAdd(&dL_dtransMat[global_id * 9 + 7],  dL_dTw.y);
				atomicAdd(&dL_dtransMat[global_id * 9 + 8],  dL_dTw.z);
			} else {
				// // Update gradients w.r.t. center of Gaussian 2D mean position
				const float dG_ddelx = -G * FilterInvSquare * d.x;
				const float dG_ddely = -G * FilterInvSquare * d.y;
				atomicAdd(&dL_dmean2D[global_id].x, dL_dG * dG_ddelx); // not scaled
				atomicAdd(&dL_dmean2D[global_id].y, dL_dG * dG_ddely); // not scaled
				atomicAdd(&dL_dtransMat[global_id * 9 + 8],  dL_dz); // propagate depth loss
			}

			// Update gradients w.r.t. opacity of the Gaussian
			atomicAdd(&(dL_dopacity[global_id*4]), G * dL_dalpha);
			atomicAdd(&(dL_dopacity[global_id*4+1]), G * dL_dalpha);
			atomicAdd(&(dL_dopacity[global_id*4+2]), G * dL_dalpha);
			atomicAdd(&(dL_dopacity[global_id*4+3]), G * dL_dalpha);

			// atomicAdd(&(dL_dopacity[global_id*4]),  L3outR.w - my_o.x);
			// atomicAdd(&(dL_dopacity[global_id*4+1]), L3outR.w - my_o.x);
			// atomicAdd(&(dL_dopacity[global_id*4+2]), L3outR.w - my_o.x);
			// atomicAdd(&(dL_dopacity[global_id*4+3]), L3outR.w - my_o.x);
			float dL_dopacity = G * dL_dalpha;





			// atomicAdd(&(dL_dopacity[global_id*4 +0]), G * dL_dalpha *  exp(-1.0f*my_rho[0]*exp_scale[0]) / 4.0f);
			// atomicAdd(&(dL_dopacity[global_id*4 +1]), G * dL_dalpha *  exp(-1.0f*my_rho[1]*exp_scale[1]) / 4.0f);
			// atomicAdd(&(dL_dopacity[global_id*4 +2]), G * dL_dalpha *  exp(-1.0f*my_rho[2]*exp_scale[2]) / 4.0f);
			// atomicAdd(&(dL_dopacity[global_id*4 +3]), G * dL_dalpha *  exp(-1.0f*my_rho[3]*exp_scale[3]) / 4.0f);

			
			glm::vec4 dL_doutput_data = glm::vec4(2.0f*dl_dc[0], 2.0f*dl_dc[1], 2.0f*dl_dc[2], dL_dopacity);
			dL_doutput_data = dL_doutput_data * dL_dSigmoid(L3out);
			glm::vec4 dL_dmiddle_data =  dL_doutput_data* output;

			glm::mat4x4 dL_doutput = glm::mat4x4(
				glm::vec4(L2outR.x*dL_doutput_data.x, L2outR.x*dL_doutput_data.y, L2outR.x*dL_doutput_data.z, L2outR.x*dL_doutput_data.w),
				glm::vec4(L2outR.y*dL_doutput_data.x, L2outR.y*dL_doutput_data.y, L2outR.y*dL_doutput_data.z, L2outR.y*dL_doutput_data.w),
				glm::vec4(L2outR.z*dL_doutput_data.x, L2outR.z*dL_doutput_data.y, L2outR.z*dL_doutput_data.z, L2outR.z*dL_doutput_data.w),
				glm::vec4(L2outR.w*dL_doutput_data.x, L2outR.w*dL_doutput_data.y, L2outR.w*dL_doutput_data.z, L2outR.w*dL_doutput_data.w)
			);
\
			glm::vec4 dL_doutput_biases = dL_doutput_data;

			dL_dmiddle_data= dL_dmiddle_data * dL_dSigmoid(L2out);

			glm::vec4 dL_dinput_data = dL_dmiddle_data * middle;
			// dL_dmiddle = dL_dmiddle_data * L1out;
			glm::mat4x4 dL_dmiddle = glm::mat4x4(
				glm::vec4(L1outR.x*dL_dmiddle_data.x, L1outR.x*dL_dmiddle_data.y, L1outR.x*dL_dmiddle_data.z, L1outR.x*dL_dmiddle_data.w),
				glm::vec4(L1outR.y*dL_dmiddle_data.x, L1outR.y*dL_dmiddle_data.y, L1outR.y*dL_dmiddle_data.z, L1outR.y*dL_dmiddle_data.w),
				glm::vec4(L1outR.z*dL_dmiddle_data.x, L1outR.z*dL_dmiddle_data.y, L1outR.z*dL_dmiddle_data.z, L1outR.z*dL_dmiddle_data.w),
				glm::vec4(L1outR.w*dL_dmiddle_data.x, L1outR.w*dL_dmiddle_data.y, L1outR.w*dL_dmiddle_data.z, L1outR.w*dL_dmiddle_data.w)
			);
			glm::vec4 dL_dmiddle_biases = dL_dmiddle_data;

		
			dL_dinput_data = dL_dinput_data * dL_dSigmoid(L1out);
			glm::vec2 dL_dsxy = dL_dinput_data * input;
			glm::mat2x4 dL_dinput = glm::mat2x4(
				glm::vec4(input_data2.x*dL_dinput_data.x, input_data2.x*dL_dinput_data.y, input_data2.x*dL_dinput_data.z, input_data2.x*dL_dinput_data.w),
				glm::vec4(input_data2.y*dL_dinput_data.x, input_data2.y*dL_dinput_data.y, input_data2.y*dL_dinput_data.z, input_data2.y*dL_dinput_data.w)
			);
			glm::vec4 dL_dinput_biases = dL_dinput_data;

			// dL_dsxy.x = dL_dsxy.x * 1.0 / (1.0 + exp(-s.x)) * (1.0 - 1.0 / (1.0 + exp(-s.x)));
			// dL_dsxy.y = dL_dsxy.y * 1.0 / (1.0 + exp(-s.y)) * (1.0 - 1.0 / (1.0 + exp(-s.y)));

			atomicAdd(&dL_dshs[102*global_id + 48 ], dL_dinput[0].x);
			atomicAdd(&dL_dshs[102*global_id + 49 ], dL_dinput[0].y);
			atomicAdd(&dL_dshs[102*global_id + 50 ], dL_dinput[0].z);
			atomicAdd(&dL_dshs[102*global_id + 51 ], dL_dinput[0].w);
			atomicAdd(&dL_dshs[102*global_id + 52 ], dL_dinput[1].x);
			atomicAdd(&dL_dshs[102*global_id + 53 ], dL_dinput[1].y);
			atomicAdd(&dL_dshs[102*global_id + 54 ], dL_dinput[1].z);
			atomicAdd(&dL_dshs[102*global_id + 55 ], dL_dinput[1].w);
			atomicAdd(&dL_dshs[102*global_id + 56 ], dL_dinput_biases.x);
			atomicAdd(&dL_dshs[102*global_id + 57 ], dL_dinput_biases.y);
			atomicAdd(&dL_dshs[102*global_id + 58 ], dL_dinput_biases.z);
			atomicAdd(&dL_dshs[102*global_id + 59 ], dL_dinput_biases.w);

			atomicAdd(&dL_dshs[102*global_id + 60 ], dL_dmiddle[0].x);
			atomicAdd(&dL_dshs[102*global_id + 61 ], dL_dmiddle[0].y);
			atomicAdd(&dL_dshs[102*global_id + 62 ], dL_dmiddle[0].z);
			atomicAdd(&dL_dshs[102*global_id + 63 ], dL_dmiddle[0].w);
			atomicAdd(&dL_dshs[102*global_id + 64 ], dL_dmiddle[1].x);
			atomicAdd(&dL_dshs[102*global_id + 65 ], dL_dmiddle[1].y);
			atomicAdd(&dL_dshs[102*global_id + 66 ], dL_dmiddle[1].z);
			atomicAdd(&dL_dshs[102*global_id + 67 ], dL_dmiddle[1].w);
			atomicAdd(&dL_dshs[102*global_id + 68 ], dL_dmiddle[2].x);
			atomicAdd(&dL_dshs[102*global_id + 69 ], dL_dmiddle[2].y);
			atomicAdd(&dL_dshs[102*global_id + 70 ], dL_dmiddle[2].z);
			atomicAdd(&dL_dshs[102*global_id + 71 ], dL_dmiddle[2].w);
			atomicAdd(&dL_dshs[102*global_id + 72 ], dL_dmiddle[3].x);
			atomicAdd(&dL_dshs[102*global_id + 73 ], dL_dmiddle[3].y);
			atomicAdd(&dL_dshs[102*global_id + 74 ], dL_dmiddle[3].z);
			atomicAdd(&dL_dshs[102*global_id + 75 ], dL_dmiddle[3].w);
			atomicAdd(&dL_dshs[102*global_id + 76 ], dL_dmiddle_biases.x);
			atomicAdd(&dL_dshs[102*global_id + 77 ], dL_dmiddle_biases.y);
			atomicAdd(&dL_dshs[102*global_id + 78 ], dL_dmiddle_biases.z);
			atomicAdd(&dL_dshs[102*global_id + 79 ], dL_dmiddle_biases.w);

			atomicAdd(&dL_dshs[102*global_id + 80 ], dL_doutput[0].x);
			atomicAdd(&dL_dshs[102*global_id + 81 ], dL_doutput[0].y);
			atomicAdd(&dL_dshs[102*global_id + 82 ], dL_doutput[0].z);
			atomicAdd(&dL_dshs[102*global_id + 83 ], dL_doutput[0].w);
			atomicAdd(&dL_dshs[102*global_id + 84 ], dL_doutput[1].x);
			atomicAdd(&dL_dshs[102*global_id + 85 ], dL_doutput[1].y);
			atomicAdd(&dL_dshs[102*global_id + 86 ], dL_doutput[1].z);
			atomicAdd(&dL_dshs[102*global_id + 87 ], dL_doutput[1].w);
			atomicAdd(&dL_dshs[102*global_id + 88 ], dL_doutput[2].x);
			atomicAdd(&dL_dshs[102*global_id + 89 ], dL_doutput[2].y);
			atomicAdd(&dL_dshs[102*global_id + 90 ], dL_doutput[2].z);
			atomicAdd(&dL_dshs[102*global_id + 91 ], dL_doutput[2].w);
			atomicAdd(&dL_dshs[102*global_id + 92 ], dL_doutput[3].x);
			atomicAdd(&dL_dshs[102*global_id + 93 ], dL_doutput[3].y);
			atomicAdd(&dL_dshs[102*global_id + 94 ], dL_doutput[3].z);
			atomicAdd(&dL_dshs[102*global_id + 95 ], dL_doutput[3].w);
			atomicAdd(&dL_dshs[102*global_id + 96 ], dL_doutput_biases.x);
			atomicAdd(&dL_dshs[102*global_id + 97 ], dL_doutput_biases.y);
			atomicAdd(&dL_dshs[102*global_id + 98 ], dL_doutput_biases.z);
			atomicAdd(&dL_dshs[102*global_id + 99 ], dL_doutput_biases.w);
			



	
				
			// dl_dtu[0] += dL_dsxy[0] * dsx_dtu[0] + dL_dsxy[1] * dsy_dtu[0];
			// dl_dtu[1] += dL_dsxy[0] * dsx_dtu[1] + dL_dsxy[1] * dsy_dtu[1];
			// dl_dtu[2] += dL_dsxy[0] * dsx_dtu[2] + dL_dsxy[1] * dsy_dtu[2];
			// dl_dtv[0] += dL_dsxy[0] * dsx_dtv[0] + dL_dsxy[1] * dsy_dtv[0];
			// dl_dtv[1] += dL_dsxy[0] * dsx_dtv[1] + dL_dsxy[1] * dsy_dtv[1];
			// dl_dtv[2] += dL_dsxy[0] * dsx_dtv[2] + dL_dsxy[1] * dsy_dtv[2];
			// dl_dtw[0] += dL_dsxy[0] * dsx_dtw[0] + dL_dsxy[1] * dsy_dtw[0];
			// dl_dtw[1] += dL_dsxy[0] * dsx_dtw[1] + dL_dsxy[1] * dsy_dtw[1];
			// dl_dtw[2] += dL_dsxy[0] * dsx_dtw[2] + dL_dsxy[1] * dsy_dtw[2];

			// 	atomicAdd(&dL_dtransMat[global_id * 9 + 0], dl_dtu[0]);
			// 	atomicAdd(&dL_dtransMat[global_id * 9 + 1], dl_dtu[1]);
			// 	atomicAdd(&dL_dtransMat[global_id * 9 + 2], dl_dtu[2]);
			// 	atomicAdd(&dL_dtransMat[global_id * 9 + 3], dl_dtv[0]);
			// 	atomicAdd(&dL_dtransMat[global_id * 9 + 4], dl_dtv[1]);
			// 	atomicAdd(&dL_dtransMat[global_id * 9 + 5], dl_dtv[2]);
			// 	atomicAdd(&dL_dtransMat[global_id * 9 + 6], dl_dtw[0]);
			// 	atomicAdd(&dL_dtransMat[global_id * 9 + 7], dl_dtw[1]);
			// 	atomicAdd(&dL_dtransMat[global_id * 9 + 8], dl_dtw[2]);

			
			
				
				// float2 dl_dmeans2d = {0.0,0.0};

				// dc_dsx[0] = Myalpha * (-1.0 * (1.0 - proj_v) * addrgb[0].x - (proj_v) * addrgb[1].x + (1.0-proj_v) * addrgb[2].x + proj_v * addrgb[3].x)   * ((sigmod_scale*exp(-sigmod_scale*d.x))/((exp(-sigmod_scale*d.x) + 1.0)*(exp(-sigmod_scale*d.x) + 1.0)));
				// dc_dsx[1] = Myalpha * (-1.0 * (1.0 - proj_v) * addrgb[0].y - (proj_v) * addrgb[1].y + (1.0-proj_v) * addrgb[2].y + proj_v * addrgb[3].y)   * ((sigmod_scale*exp(-sigmod_scale*d.x))/((exp(-sigmod_scale*d.x) + 1.0)*(exp(-sigmod_scale*d.x) + 1.0)));
				// dc_dsx[2] = Myalpha * (-1.0 * (1.0 - proj_v) * addrgb[0].z - (proj_v) * addrgb[1].z + (1.0-proj_v) * addrgb[2].z + proj_v * addrgb[3].z)   * ((sigmod_scale*exp(-sigmod_scale*d.x))/((exp(-sigmod_scale*d.x) + 1.0)*(exp(-sigmod_scale*d.x) + 1.0)));
				// dc_dsy[0] = Myalpha * (-1.0 * (1.0 - proj_u) * addrgb[0].x + (1.0 - proj_u) * addrgb[1].x - (proj_u) * addrgb[2].x + proj_u * addrgb[3].x) * ((sigmod_scale*exp(-sigmod_scale*d.y))/((exp(-sigmod_scale*d.y) + 1.0)*(exp(-sigmod_scale*d.y) + 1.0)));
				// dc_dsy[1] = Myalpha * (-1.0 * (1.0 - proj_u) * addrgb[0].y + (1.0 - proj_u) * addrgb[1].y - (proj_u) * addrgb[2].y + proj_u * addrgb[3].y) * ((sigmod_scale*exp(-sigmod_scale*d.y))/((exp(-sigmod_scale*d.y) + 1.0)*(exp(-sigmod_scale*d.y) + 1.0)));
				// dc_dsy[2] = Myalpha * (-1.0 * (1.0 - proj_u) * addrgb[0].z + (1.0 - proj_u) * addrgb[1].z - (proj_u) * addrgb[2].z + proj_u * addrgb[3].z) * ((sigmod_scale*exp(-sigmod_scale*d.y))/((exp(-sigmod_scale*d.y) + 1.0)*(exp(-sigmod_scale*d.y) + 1.0)));

				// dop_dsx = (-1.0 * (1.0 - proj_v) * my_o.x - (proj_v) * my_o.y + (1.0-proj_v) * my_o.z + proj_v * my_o.w)   * ((sigmod_scale*exp(-sigmod_scale*d.x))/((exp(-sigmod_scale*d.x) + 1.0)*(exp(-sigmod_scale*d.x) + 1.0)));
				// dop_dsy = (-1.0 * (1.0 - proj_u) * my_o.x + (1.0 - proj_u) * my_o.y - (proj_u) * my_o.z + proj_u * my_o.w) * ((sigmod_scale*exp(-sigmod_scale*d.y))/((exp(-sigmod_scale*d.y) + 1.0)*(exp(-sigmod_scale*d.y) + 1.0)));
				// for (int ch = 0; ch < C; ch++)
				// {
				// 	dl_dmeans2d.x += dl_dc[ch] * dc_dsx[ch];
				// 	dl_dmeans2d.y += dl_dc[ch] * dc_dsy[ch];
				// }
				// dl_dmeans2d.x += G * dL_dalpha * dop_dsx;
				// dl_dmeans2d.y += G * dL_dalpha * dop_dsy;

				// atomicAdd(&dL_dmean2D[global_id].x, dl_dmeans2d.x);
				// atomicAdd(&dL_dmean2D[global_id].y, dl_dmeans2d.y); 
				
				
			

		


		}
	}
}


__device__ void compute_transmat_aabb(
	int idx, 
	const float* Ts_precomp,
	const float3* p_origs, 
	const glm::vec2* scales, 
	const glm::vec4* rots, 
	const float* projmatrix, 
	const float* viewmatrix, 
	const int W, const int H, 
	const float3* dL_dnormals,
	const float3* dL_dmean2Ds, 
	float* dL_dTs, 
	glm::vec3* dL_dmeans, 
	glm::vec2* dL_dscales,
	 glm::vec4* dL_drots)
{
	glm::mat3 T;
	float3 normal;
	glm::mat3x4 P;
	glm::mat3 R;
	glm::mat3 S;
	float3 p_orig;
	glm::vec4 rot;
	glm::vec2 scale;
	
	// Get transformation matrix of the Gaussian
	if (Ts_precomp != nullptr) {
		T = glm::mat3(
			Ts_precomp[idx * 9 + 0], Ts_precomp[idx * 9 + 1], Ts_precomp[idx * 9 + 2],
			Ts_precomp[idx * 9 + 3], Ts_precomp[idx * 9 + 4], Ts_precomp[idx * 9 + 5],
			Ts_precomp[idx * 9 + 6], Ts_precomp[idx * 9 + 7], Ts_precomp[idx * 9 + 8]
		);
		normal = {0.0, 0.0, 0.0};
	} else {
		p_orig = p_origs[idx];
		rot = rots[idx];
		scale = scales[idx];
		R = quat_to_rotmat(rot);
		S = scale_to_mat(scale, 1.0f);
		
		glm::mat3 L = R * S;
		glm::mat3x4 M = glm::mat3x4(
			glm::vec4(L[0], 0.0),
			glm::vec4(L[1], 0.0),
			glm::vec4(p_orig.x, p_orig.y, p_orig.z, 1)
		);

		glm::mat4 world2ndc = glm::mat4(
			projmatrix[0], projmatrix[4], projmatrix[8], projmatrix[12],
			projmatrix[1], projmatrix[5], projmatrix[9], projmatrix[13],
			projmatrix[2], projmatrix[6], projmatrix[10], projmatrix[14],
			projmatrix[3], projmatrix[7], projmatrix[11], projmatrix[15]
		);

		glm::mat3x4 ndc2pix = glm::mat3x4(
			glm::vec4(float(W) / 2.0, 0.0, 0.0, float(W-1) / 2.0),
			glm::vec4(0.0, float(H) / 2.0, 0.0, float(H-1) / 2.0),
			glm::vec4(0.0, 0.0, 0.0, 1.0)
		);

		P = world2ndc * ndc2pix;
		T = glm::transpose(M) * P;
		normal = transformVec4x3({L[2].x, L[2].y, L[2].z}, viewmatrix);
	}

	// Update gradients w.r.t. transformation matrix of the Gaussian
	glm::mat3 dL_dT = glm::mat3(
		dL_dTs[idx*9+0], dL_dTs[idx*9+1], dL_dTs[idx*9+2],
		dL_dTs[idx*9+3], dL_dTs[idx*9+4], dL_dTs[idx*9+5],
		dL_dTs[idx*9+6], dL_dTs[idx*9+7], dL_dTs[idx*9+8]
	);
	float3 dL_dmean2D = dL_dmean2Ds[idx];
	if(dL_dmean2D.x != 0 || dL_dmean2D.y != 0)
	{
		glm::vec3 t_vec = glm::vec3(9.0f, 9.0f, -1.0f);
		float d = glm::dot(t_vec, T[2] * T[2]);
		glm::vec3 f_vec = t_vec * (1.0f / d);
		glm::vec3 dL_dT0 = dL_dmean2D.x * f_vec * T[2];
		glm::vec3 dL_dT1 = dL_dmean2D.y * f_vec * T[2];
		glm::vec3 dL_dT3 = dL_dmean2D.x * f_vec * T[0] + dL_dmean2D.y * f_vec * T[1];
		glm::vec3 dL_df = dL_dmean2D.x * T[0] * T[2] + dL_dmean2D.y * T[1] * T[2];
		float dL_dd = glm::dot(dL_df, f_vec) * (-1.0 / d);
		glm::vec3 dd_dT3 = t_vec * T[2] * 2.0f;
		dL_dT3 += dL_dd * dd_dT3;
		dL_dT[0] += dL_dT0;
		dL_dT[1] += dL_dT1;
		dL_dT[2] += dL_dT3;

		if (Ts_precomp != nullptr) {
			dL_dTs[idx * 9 + 0] = dL_dT[0].x;
			dL_dTs[idx * 9 + 1] = dL_dT[0].y;
			dL_dTs[idx * 9 + 2] = dL_dT[0].z;
			dL_dTs[idx * 9 + 3] = dL_dT[1].x;
			dL_dTs[idx * 9 + 4] = dL_dT[1].y;
			dL_dTs[idx * 9 + 5] = dL_dT[1].z;
			dL_dTs[idx * 9 + 6] = dL_dT[2].x;
			dL_dTs[idx * 9 + 7] = dL_dT[2].y;
			dL_dTs[idx * 9 + 8] = dL_dT[2].z;
			return;
		}
	}
	
	if (Ts_precomp != nullptr) return;

	// Update gradients w.r.t. scaling, rotation, position of the Gaussian
	glm::mat3x4 dL_dM = P * glm::transpose(dL_dT);
	float3 dL_dtn = transformVec4x3Transpose(dL_dnormals[idx], viewmatrix);
#if DUAL_VISIABLE
	float3 p_view = transformPoint4x3(p_orig, viewmatrix);
	float cos = -sumf3(p_view * normal);
	float multiplier = cos > 0 ? 1: -1;
	dL_dtn = multiplier * dL_dtn;
#endif
	glm::mat3 dL_dRS = glm::mat3(
		glm::vec3(dL_dM[0]),
		glm::vec3(dL_dM[1]),
		glm::vec3(dL_dtn.x, dL_dtn.y, dL_dtn.z)
	);

	glm::mat3 dL_dR = glm::mat3(
		dL_dRS[0] * glm::vec3(scale.x),
		dL_dRS[1] * glm::vec3(scale.y),
		dL_dRS[2]);
	
	dL_drots[idx] = quat_to_rotmat_vjp(rot, dL_dR);
	dL_dscales[idx] = glm::vec2(
		(float)glm::dot(dL_dRS[0], R[0]),
		(float)glm::dot(dL_dRS[1], R[1])
	);
	dL_dmeans[idx] = glm::vec3(dL_dM[2]);
}

template<int C>
__global__ void preprocessCUDA(
	int P, int D, int M,
	const float3* means3D,
	const float* transMats,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec2* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, 
	const float focal_y,
	const float tan_fovx,
	const float tan_fovy,
	const glm::vec3* campos, 
	// grad input
	float* dL_dtransMats,
	const float* dL_dnormal3Ds,
	float* dL_dcolors,
	float* dL_dshs,
	float3* dL_dmean2Ds,
	glm::vec3* dL_dmean3Ds,
	glm::vec2* dL_dscales,
	glm::vec4* dL_drots,
	float2* wichzone)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P || !(radii[idx] > 0))
		return;

	const int W = int(focal_x * tan_fovx * 2);
	const int H = int(focal_y * tan_fovy * 2);
	const float * Ts_precomp = (scales) ? nullptr : transMats;
	compute_transmat_aabb(
		idx, 
		Ts_precomp,
		means3D, scales, rotations, 
		projmatrix, viewmatrix, W, H, 
		(float3*)dL_dnormal3Ds, 
		dL_dmean2Ds,
		(dL_dtransMats), 
		dL_dmean3Ds, 
		dL_dscales, 
		dL_drots
	);

	if (shs)
		computeColorFromSH(idx, D, M, (glm::vec3*)means3D, *campos, shs, clamped, (glm::vec3*)dL_dcolors, (glm::vec3*)dL_dmean3Ds, (glm::vec3*)dL_dshs, wichzone);
	
	// hack the gradient here for densitification
	float depth = transMats[idx * 9 + 8];
	dL_dmean2Ds[idx].x = dL_dtransMats[idx * 9 + 2] * depth * 0.5 * float(W); // to ndc 
	dL_dmean2Ds[idx].y = dL_dtransMats[idx * 9 + 5] * depth * 0.5 * float(H); // to ndc
}


void BACKWARD::preprocess(
	int P, int D, int M,
	const float3* means3D,
	const int* radii,
	const float* shs,
	const bool* clamped,
	const glm::vec2* scales,
	const glm::vec4* rotations,
	const float scale_modifier,
	const float* transMats,
	const float* viewmatrix,
	const float* projmatrix,
	const float focal_x, const float focal_y,
	const float tan_fovx, const float tan_fovy,
	const glm::vec3* campos, 
	float3* dL_dmean2Ds,
	const float* dL_dnormal3Ds,
	float* dL_dtransMats,
	float* dL_dcolors,
	float* dL_dshs,
	glm::vec3* dL_dmean3Ds,
	glm::vec2* dL_dscales,
	glm::vec4* dL_drots,
	float2* wichzone)
{	
	preprocessCUDA<NUM_CHANNELS><< <(P + 255) / 256, 256 >> > (
		P, D, M,
		(float3*)means3D,
		transMats,
		radii,
		shs,
		clamped,
		(glm::vec2*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		viewmatrix,
		projmatrix,
		focal_x, 
		focal_y,
		tan_fovx,
		tan_fovy,
		campos,	
		dL_dtransMats,
		dL_dnormal3Ds,
		dL_dcolors,
		dL_dshs,
		dL_dmean2Ds,
		dL_dmean3Ds,
		dL_dscales,
		dL_drots,
		wichzone
	);
}

void BACKWARD::render(
	const dim3 grid, const dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float* bg_color,
	const float2* means2D,
	const float4* normal_opacity,
	const float4* my_opacity,
	const float* colors,
	const float* transMats,
	const float* depths,
	const float* final_Ts,
	const uint32_t* n_contrib,
	const float* dL_dpixels,
	const float* dL_depths,
	float * dL_dtransMat,
	float3* dL_dmean2D,
	float* dL_dnormal3D,
	float* dL_dopacity,
	float* dL_dcolors,
	const float* shs,
	float2* wichzone,
	float* dL_dshs)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> >(
		ranges,
		point_list,
		W, H,
		focal_x, focal_y,
		bg_color,
		means2D,
		normal_opacity,
		my_opacity,
		transMats,
		colors,
		depths,
		final_Ts,
		n_contrib,
		dL_dpixels,
		dL_depths,
		dL_dtransMat,
		dL_dmean2D,
		dL_dnormal3D,
		dL_dopacity,
		dL_dcolors,
		shs,
		wichzone,
		dL_dshs
		);
}
